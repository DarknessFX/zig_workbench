
#include <hip/hip_runtime.h>
#include <stdio.h>


// DEVICE
extern "C" __global__ void _helloWorld(int arg1, float arg2) {
  printf("CUDA Function called with \narg1: %d\narg2: %.2f\n", arg1, arg2);
}

// HOST
// Helper function to be able to call << >> commands.
extern "C" __declspec(dllexport) void helloWorld(int arg1, float arg2) {
  _helloWorld<<<1, 1>>>(arg1, arg2);
  hipDeviceSynchronize();
}